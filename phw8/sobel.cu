
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>
#include<math.h>

#define n 192

#define input_file  "phw8.raw"
#define output_file "output.raw"

__global__ void sobel(unsigned char *a, unsigned char *c) {
	int myx, myy, i;
	myx = blockIdx.x * blockDim.x + threadIdx.x;
	myy = blockIdx.y * blockDim.y + threadIdx.y;    
    
    int gx[3][3] =
    {
        {-1,2,-1},
        {0,0,0},
        {1,2,1}
    };
    
    int gy[3][3] =
    {
        {-1,0,1},
        {-2,0,2},
        {-1,0,1}
    };
    
    int dx = 0, dy = 0;

    dx += (a[((myx-1)*(n+2)) + myy-1] * gx[0][0])
       +  (a[((myx-1)*(n+2)) + myy] * gx[0][1])
       +  (a[((myx-1)*(n+2)) + myy+1] * gx[0][2])
       
       +  (a[((myx)*(n+2)) + myy-1] * gx[1][0])
       +  (a[((myx)*(n+2)) + myy] * gx[1][1])
       +  (a[((myx)*(n+2)) + myy+1] * gx[1][2])
       
       +  (a[((myx+1)*(n+2)) + myy-1] * gx[2][0])
       +  (a[((myx+1)*(n+2)) + myy] * gx[2][1])
       +  (a[((myx+1)*(n+2)) + myy+1] * gx[2][2]);
    
    dy += (a[((myx-1)*(n+2)) + myy-1] * gy[0][0])
       +  (a[((myx-1)*(n+2)) + myy] * gy[0][1])
       +  (a[((myx-1)*(n+2)) + myy+1] * gy[0][2])
       
       +  (a[((myx)*(n+2)) + myy-1] * gy[1][0])
       +  (a[((myx)*(n+2)) + myy] * gy[1][1])
       +  (a[((myx)*(n+2)) + myy+1] * gy[1][2])
       
       +  (a[((myx+1)*(n+2)) + myy-1] * gy[2][0])
       +  (a[((myx+1)*(n+2)) + myy] * gy[2][1])
       +  (a[((myx+1)*(n+2)) + myy+1] * gy[2][2]);
    
    
	c[myx*n+myy] = (unsigned char)sqrt( ( ((double)dx)*((double)dx) + ((double)dy)*((double)dy) ) );
}

int main() {
	unsigned char *a = (unsigned char*)malloc(sizeof(unsigned char)*(n+2)*(n+2));
	unsigned char *c = (unsigned char*)malloc(sizeof(unsigned char)*n*n);
    
	hipEvent_t start, stop;
	float time;
    
    FILE *fp;
	
    dim3 numBlocks(64,64);
    dim3 threadsPerBlock(3,3);
    
    
    // Read image
    if (!(fp=fopen(input_file, "rb"))) {
		printf("can not opern file\n");
		return 1;
	}
	fread(a, sizeof(unsigned char), (n+2)*(n+2), fp);
	fclose(fp);


	unsigned char *gpua, *gpuc;

	hipMalloc((void**)&gpua, sizeof(unsigned char *)*(n+2)*(n+2));
	hipMalloc((void**)&gpuc, sizeof(unsigned char *)*n*n);

	hipMemcpy(gpua, a, sizeof(unsigned char *)*(n+2)*(n+2), hipMemcpyHostToDevice);
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);
	sobel<<<numBlocks, threadsPerBlock>>> (gpua, gpuc);
	hipEventRecord(stop,0);


	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipMemcpy(c, gpuc, sizeof(unsigned char *)*n*n, hipMemcpyDeviceToHost);

    if (!(fp=fopen(output_file,"wb"))) {
		printf("can not opern file\n");
		return 1;
	}	
	fwrite(c, sizeof(unsigned char),n*n, fp);
    fclose(fp);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(a);
	free(c);

	hipFree(gpua);
	hipFree(gpuc);

	return 0;

}
